#include "hip/hip_runtime.h"
// Copyright (C) 2021 Athena Elafrou (ae488@cam.ac.uk)
// SPDX-License-Identifier:    MIT

#include "L2GMap.h"
#include "Matrix.h"
#include "cg_cuda.h"
#include "hip/hip_runtime_api.h"

#include <hipblas.h>

//-----------------------------------------------------------------------------
__global__ void compute_alpha(double* rnorm_old, double* pdotAp, double* alpha,
                              double* neg_alpha)
{
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid == 0) {
    alpha[0] = (rnorm_old[0] * rnorm_old[0]) / pdotAp[0];
    neg_alpha[0] = -(alpha[0]);
  }
}
//-----------------------------------------------------------------------------
__global__ void compute_beta(double* rnorm_old, double* rnorm_new, double* beta)
{
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid == 0) {
    beta[0] = (rnorm_new[0] * rnorm_new[0]) / (rnorm_old[0] * rnorm_old[0]);
  }
}
//-----------------------------------------------------------------------------
__global__ void compute_sqrt(double* in, double* out)
{
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid == 0) {
    out[0] = sqrt(in[0]);
  }
}
//-----------------------------------------------------------------------------
std::tuple<double*, int> spmv::cg(MPI_Comm comm, const spmv::Matrix<double>& A,
                                  double* b, int kmax, double rtol)
{
  int mpi_rank;
  MPI_Comm_rank(comm, &mpi_rank);

  std::shared_ptr<const spmv::L2GMap> col_l2g = A.col_map();
  std::shared_ptr<const spmv::L2GMap> row_l2g = A.row_map();

  // Check the row map is unghosted
  if (row_l2g->num_ghosts() > 0)
    throw std::runtime_error("spmv::cg - Error: A.row_map() has ghost entries");

  int M = row_l2g->local_size();
  int N_padded = col_l2g->local_size() + col_l2g->num_ghosts();

  // Create CUDA streams to offload operations
  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);

  // Get handle to the CUBLAS context
  hipblasHandle_t cublas_handle = 0;
  CHECK_CUBLAS(hipblasCreate(&cublas_handle));
  CHECK_CUBLAS(hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_DEVICE));

  // Allocate device pointers for vectors
  double* d_x = nullptr;
  double* d_p = nullptr;
  double* d_r = nullptr;
  double* d_Ap = nullptr;

  CHECK_CUDA(hipMalloc((void**)&d_x, N_padded * sizeof(double)));
  CHECK_CUDA(hipMalloc((void**)&d_p, N_padded * sizeof(double)));
  CHECK_CUDA(hipMalloc((void**)&d_r, M * sizeof(double)));
  CHECK_CUDA(hipMalloc((void**)&d_Ap, M * sizeof(double)));
  // FIXME if there are more DMA engines per direction maybe overlap
  // Use asynchronous memory copies to hide launch overheads
  CHECK_CUDA(hipMemcpyAsync(d_p, b, M * sizeof(double), hipMemcpyHostToDevice,
                             stream1));
  CHECK_CUDA(hipMemcpyAsync(d_r, d_p, M * sizeof(double),
                             hipMemcpyDeviceToDevice, stream1));

  double* d_alpha = nullptr;
  CHECK_CUDA(hipMalloc((void**)&d_alpha, sizeof(double)));
  double* d_neg_alpha = nullptr;
  CHECK_CUDA(hipMalloc((void**)&d_neg_alpha, sizeof(double)));
  double* d_beta = nullptr;
  CHECK_CUDA(hipMalloc((void**)&d_beta, sizeof(double)));

  double* d_rnorm0 = nullptr;
  CHECK_CUDA(hipMalloc((void**)&d_rnorm0, sizeof(double)));
  double* d_rnorm_old = nullptr;
  CHECK_CUDA(hipMalloc((void**)&d_rnorm_old, sizeof(double)));
  double* d_rnorm_new = nullptr;
  CHECK_CUDA(hipMalloc((void**)&d_rnorm_new, sizeof(double)));
  double* d_rnorm_local = nullptr;
  CHECK_CUDA(hipMalloc((void**)&d_rnorm_local, sizeof(double)));

  double* d_pdotAp_local = nullptr;
  CHECK_CUDA(hipMalloc((void**)&d_pdotAp_local, sizeof(double)));
  double* d_pdotAp = nullptr;
  CHECK_CUDA(hipMalloc((void**)&d_pdotAp, sizeof(double)));

  double scalar_one = 1;
  double* d_scalar_one = nullptr;
  CHECK_CUDA(hipMalloc((void**)&d_scalar_one, sizeof(double)));
  CHECK_CUDA(hipMemcpyAsync(d_scalar_one, &scalar_one, sizeof(double),
                             hipMemcpyHostToDevice, stream1));

  CHECK_CUBLAS(hipblasSetStream(cublas_handle, stream1));
  CHECK_CUBLAS(hipblasDdot(cublas_handle, M, d_r, 1, d_r, 1, d_rnorm_local));
  hipStreamSynchronize(stream1);
  MPI_Allreduce(d_rnorm_local, d_rnorm0, 1, MPI_DOUBLE, MPI_SUM, comm);
  double rnorm0;
  CHECK_CUDA(
      hipMemcpy(&rnorm0, d_rnorm0, sizeof(double), hipMemcpyDeviceToHost));
  rnorm0 = sqrt(rnorm0);
  CHECK_CUDA(
      hipMemcpy(d_rnorm_old, &rnorm0, sizeof(double), hipMemcpyHostToDevice));

  // Iterations of CG
  int k = 0;
  hipEvent_t event;
  hipEventCreateWithFlags(&event, hipEventDisableTiming);
  while (k < kmax) {
    ++k;

    // Ap = A.p
    col_l2g->update(d_p, stream1);
    A.mult(d_p, d_Ap, stream1);

    // Calculate alpha = r.r/p.Ap
    CHECK_CUBLAS(hipblasDdot(cublas_handle, M, d_p, 1, d_Ap, 1, d_pdotAp_local));
    CHECK_CUDA(hipStreamSynchronize(stream1));
    MPI_Allreduce(d_pdotAp_local, d_pdotAp, 1, MPI_DOUBLE, MPI_SUM, comm);
    compute_alpha<<<1, 1, 0, stream1>>>(d_rnorm_old, d_pdotAp, d_alpha,
                                        d_neg_alpha);
    hipEventRecord(event, stream1);

    // Update x and r
    // These operations can be done in parallel, so launch in seperate streams
    // x = x + alpha*p
    CHECK_CUDA(hipStreamWaitEvent(stream2, event));
    CHECK_CUBLAS(hipblasSetStream(cublas_handle, stream2));
    CHECK_CUBLAS(hipblasDaxpy(cublas_handle, M, d_alpha, d_p, 1, d_x, 1));
    // r = r - alpha*Ap
    CHECK_CUBLAS(hipblasSetStream(cublas_handle, stream1));
    CHECK_CUBLAS(hipblasDaxpy(cublas_handle, M, d_neg_alpha, d_Ap, 1, d_r, 1));

    // Update rnorm
    CHECK_CUBLAS(hipblasDdot(cublas_handle, M, d_r, 1, d_r, 1, d_rnorm_local));
    hipStreamSynchronize(stream1);
    MPI_Allreduce(d_rnorm_local, d_rnorm_new, 1, MPI_DOUBLE, MPI_SUM, comm);
    compute_sqrt<<<1, 1, 0, stream1>>>(d_rnorm_new, d_rnorm_new);
    compute_beta<<<1, 1, 0, stream1>>>(d_rnorm_old, d_rnorm_new, d_beta);
    // FIXME Can you hide this?
    CHECK_CUDA(hipMemcpyAsync(d_rnorm_old, d_rnorm_new, sizeof(double),
                               hipMemcpyDeviceToDevice, stream1));

    double rnorm_new;
    CHECK_CUDA(hipMemcpyAsync(&rnorm_new, d_rnorm_new, sizeof(double),
                               hipMemcpyDeviceToHost, stream1));
    hipStreamSynchronize(stream1);
    if (rnorm_new / rnorm0 < rtol)
      break;

    // Update p.
    // p = r + beta*p
    CHECK_CUBLAS(hipblasDscal(cublas_handle, M, d_beta, d_p, 1));
    CHECK_CUBLAS(hipblasDaxpy(cublas_handle, M, d_scalar_one, d_r, 1, d_p, 1));
  }

  // Cleanup
  CHECK_CUDA(hipStreamDestroy(stream1));
  CHECK_CUDA(hipStreamDestroy(stream2));
  CHECK_CUDA(hipFree(d_Ap));
  CHECK_CUDA(hipFree(d_p));
  CHECK_CUDA(hipFree(d_r));
  CHECK_CUDA(hipFree(d_scalar_one));
  CHECK_CUDA(hipFree(d_alpha));
  CHECK_CUDA(hipFree(d_neg_alpha));
  CHECK_CUDA(hipFree(d_beta));
  CHECK_CUDA(hipFree(d_rnorm_old));
  CHECK_CUDA(hipFree(d_rnorm_new));
  CHECK_CUDA(hipFree(d_rnorm_local));
  CHECK_CUDA(hipFree(d_pdotAp_local));
  CHECK_CUDA(hipFree(d_pdotAp));
  CHECK_CUBLAS(hipblasDestroy(cublas_handle));

  return std::make_tuple(d_x, k);
}
//-----------------------------------------------------------------------------

// hipMallocAsync not supported yet by UCX
// std::tuple<double*, int> spmv::cg(MPI_Comm comm, const spmv::Matrix<double>&
// A, double* b, int kmax, double rtol)
// {
//   int mpi_rank;
//   MPI_Comm_rank(comm, &mpi_rank);

//   std::shared_ptr<const spmv::L2GMap> col_l2g = A.col_map();
//   std::shared_ptr<const spmv::L2GMap> row_l2g = A.row_map();

//   // Check the row map is unghosted
//   if (row_l2g->num_ghosts() > 0)
//     throw std::runtime_error("spmv::cg - Error: A.row_map() has ghost
//     entries");

//   int M = row_l2g->local_size();
//   int N_padded = col_l2g->local_size() + col_l2g->num_ghosts();

//   // Create CUDA stream to offload operations
//   hipStream_t stream;
//   hipStreamCreate(&stream);

//   // Get handle to the CUBLAS context
//   hipblasHandle_t cublas_handle = 0;
//   CHECK_CUBLAS(hipblasCreate(&cublas_handle));
//   CHECK_CUBLAS(hipblasSetStream(cublas_handle, stream));
//   CHECK_CUBLAS(hipblasSetPointerMode(cublas_handle,
//   HIPBLAS_POINTER_MODE_DEVICE));

//   // Allocate device pointers for vectors
//   double* d_x = nullptr;
//   double* d_p = nullptr;
//   double* d_r = nullptr;
//   double* d_Ap = nullptr;
//   CHECK_CUDA(hipMallocAsync((void **)&d_x, N_padded * sizeof(double),
//   stream)); CHECK_CUDA(hipMalloc((void **)&d_p, N_padded * sizeof(double)));
//   CHECK_CUDA(hipMallocAsync((void **)&d_r, M * sizeof(double), stream));
//   CHECK_CUDA(hipMallocAsync((void **)&d_Ap, M * sizeof(double), stream));
//   CHECK_CUDA(hipMemcpyAsync(d_p, b, M * sizeof(double),
//   hipMemcpyHostToDevice, stream)); CHECK_CUDA(hipMemcpyAsync(d_r, d_p, M *
//   sizeof(double), hipMemcpyDeviceToDevice, stream));

//   double* d_alpha = nullptr;
//   CHECK_CUDA(hipMallocAsync((void **)&d_alpha, sizeof(double), stream));
//   double* d_neg_alpha = nullptr;
//   CHECK_CUDA(hipMallocAsync((void **)&d_neg_alpha, sizeof(double), stream));
//   double* d_beta = nullptr;
//   CHECK_CUDA(hipMallocAsync((void **)&d_beta, sizeof(double), stream));

//   double* d_rnorm0 = nullptr;
//   CHECK_CUDA(hipMalloc((void **)&d_rnorm0, sizeof(double)));
//   double* d_rnorm_old = nullptr;
//   CHECK_CUDA(hipMallocAsync((void **)&d_rnorm_old, sizeof(double), stream));
//   double* d_rnorm_new = nullptr;
//   CHECK_CUDA(hipMalloc((void **)&d_rnorm_new, sizeof(double)));
//   double* d_rnorm_local = nullptr;
//   CHECK_CUDA(hipMalloc((void **)&d_rnorm_local, sizeof(double)));

//   double* d_pdotAp_local = nullptr;
//   CHECK_CUDA(hipMalloc((void **)&d_pdotAp_local, sizeof(double)));
//   double* d_pdotAp = nullptr;
//   CHECK_CUDA(hipMalloc((void **)&d_pdotAp, sizeof(double)));

//   double scalar_one = 1;
//   double* d_scalar_one = nullptr;
//   CHECK_CUDA(hipMallocAsync((void **)&d_scalar_one, sizeof(double),
//   stream)); CHECK_CUDA(hipMemcpyAsync(d_scalar_one, &scalar_one,
//   sizeof(double), hipMemcpyHostToDevice, stream));

//   int k = 0;
//   CHECK_CUBLAS(hipblasDdot(cublas_handle, M, d_r, 1, d_r, 1, d_rnorm_local));
//   hipStreamSynchronize(stream);
//   MPI_Allreduce(d_rnorm_local, d_rnorm0, 1, MPI_DOUBLE, MPI_SUM, comm);
//   double rnorm0;
//   CHECK_CUDA(hipMemcpy(&rnorm0, d_rnorm0, sizeof(double),
//   hipMemcpyDeviceToHost)); rnorm0 = sqrt(rnorm0);

//   // Iterations of CG
//   CHECK_CUDA(hipMemcpy(d_rnorm_old, &rnorm0, sizeof(double),
//   hipMemcpyHostToDevice)); while (k < kmax) {
//     ++k;

//     // Ap = A.p
//     //col_l2g->update(d_p, stream);
//     A.mult(d_p, d_Ap, stream);

//     // Calculate alpha = r.r/p.Ap
//     CHECK_CUBLAS(hipblasDdot(cublas_handle, M, d_p, 1, d_Ap, 1,
//     d_pdotAp_local)); hipStreamSynchronize(stream);
//     MPI_Allreduce(d_pdotAp_local, d_pdotAp, 1, MPI_DOUBLE, MPI_SUM, comm);
//     compute_alpha<<<1, 1, 0, stream>>>(d_rnorm_old, d_pdotAp, d_alpha,
//     d_neg_alpha);

//     // Update x and r
//     // x = x + alpha*p
//     CHECK_CUBLAS(hipblasDaxpy(cublas_handle, M, d_alpha, d_p, 1, d_x, 1));
//     // r = r - alpha*Ap
//     CHECK_CUBLAS(hipblasDaxpy(cublas_handle, M, d_neg_alpha, d_Ap, 1, d_r,
//     1));

//     // Update rnorm
//     CHECK_CUBLAS(hipblasDdot(cublas_handle, M, d_r, 1, d_r, 1,
//     d_rnorm_local)); hipStreamSynchronize(stream);
//     MPI_Allreduce(d_rnorm_local, d_rnorm_new, 1, MPI_DOUBLE, MPI_SUM, comm);
//     compute_sqrt<<<1, 1, 0, stream>>>(d_rnorm_new, d_rnorm_new);
//     compute_beta<<<1, 1, 0, stream>>>(d_rnorm_old, d_rnorm_new, d_beta);
//     CHECK_CUDA(hipMemcpyAsync(d_rnorm_old, d_rnorm_new, sizeof(double),
//     hipMemcpyDeviceToDevice, stream));

//     double rnorm_new;
//     CHECK_CUDA(hipMemcpy(&rnorm_new, d_rnorm_new, sizeof(double),
//     hipMemcpyDeviceToHost)); if (rnorm_new / rnorm0 < rtol)
//       break;

//     // Update p.
//     // p = r + beta*p
//     CHECK_CUBLAS(hipblasDscal(cublas_handle, M, d_beta, d_p, 1));
//     CHECK_CUBLAS(hipblasDaxpy(cublas_handle, M, d_scalar_one, d_r, 1, d_p,
//     1));
//   }

//   // Cleanup
//   CHECK_CUBLAS(hipblasDestroy(cublas_handle));
//   CHECK_CUDA(hipFreeAsync(d_Ap, stream));
//   CHECK_CUDA(hipFreeAsync(d_p, stream));
//   CHECK_CUDA(hipFreeAsync(d_r, stream));
//   CHECK_CUDA(hipFreeAsync(d_scalar_one, stream));
//   CHECK_CUDA(hipFreeAsync(d_alpha, stream));
//   CHECK_CUDA(hipFreeAsync(d_neg_alpha, stream));
//   CHECK_CUDA(hipFreeAsync(d_beta, stream));
//   CHECK_CUDA(hipFreeAsync(d_rnorm_old, stream));
//   CHECK_CUDA(hipFreeAsync(d_rnorm_new, stream));
//   CHECK_CUDA(hipFreeAsync(d_rnorm_local, stream));
//   CHECK_CUDA(hipFreeAsync(d_pdotAp_local, stream));
//   CHECK_CUDA(hipFreeAsync(d_pdotAp, stream));
//   CHECK_CUDA(hipStreamDestroy(stream));

//   return std::make_tuple(d_x, k);
// }
