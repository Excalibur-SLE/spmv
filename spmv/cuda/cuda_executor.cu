#include "hip/hip_runtime.h"
// Copyright (C) 2022 Athena Elafrou (ae488@cam.ac.uk)
// SPDX-License-Identifier:    MIT

#include "coo_matrix.h"
#include "csr_matrix.h"
#include "cuda_executor.h"
#include "reference_executor.h"

namespace spmv
{

CudaExecutor::CudaExecutor(int device_id, std::shared_ptr<DeviceExecutor> host)
    : _host(host)
{
  CHECK_CUDA(hipSetDevice(device_id));
  this->_dev_info.device_id = device_id;
  CHECK_CUBLAS(hipblasCreate(&_cublas_handle));
  CHECK_CUSPARSE(hipsparseCreate(&_cusparse_handle));
}

CudaExecutor::~CudaExecutor()
{
  CHECK_CUSPARSE(hipsparseDestroy(_cusparse_handle));
  CHECK_CUBLAS(hipblasDestroy(_cublas_handle));
}

void CudaExecutor::synchronize() const
{
  if (_stream) {
    CHECK_CUDA(hipStreamSynchronize(_stream));
  } else {
    CHECK_CUDA(hipDeviceSynchronize());
  }
}

int CudaExecutor::get_num_devices() const
{
  int num_devices = 0;
  CHECK_CUDA(hipGetDeviceCount(&num_devices));
  return num_devices;
}

int CudaExecutor::get_num_cus() const
{
  hipDeviceProp_t device_prop;
  CHECK_CUDA(
      hipGetDeviceProperties(&device_prop, this->get_device_info().device_id));
  return device_prop.multiProcessorCount;
}

void* CudaExecutor::_alloc(size_t num_bytes) const
{
  void* ptr = nullptr;
  CHECK_CUDA(hipMalloc((void**)&ptr, num_bytes));
  return ptr;
}

void CudaExecutor::_free(void* ptr) const { CHECK_CUDA(hipFree(ptr)); }

void CudaExecutor::_memset(void* ptr, int value, size_t num_bytes) const
{
  CHECK_CUDA(hipMemset(ptr, value, num_bytes));
};

void CudaExecutor::_copy(void* dst_ptr, const void* src_ptr,
                         size_t num_bytes) const
{
  CHECK_CUDA(hipMemcpy(dst_ptr, src_ptr, num_bytes, hipMemcpyDeviceToDevice));
}

void CudaExecutor::_copy_async(void* dst_ptr, const void* src_ptr,
                               size_t num_bytes, void* obj) const
{
  hipStream_t* stream = reinterpret_cast<hipStream_t*>(obj);
  CHECK_CUDA(hipMemcpyAsync(dst_ptr, src_ptr, num_bytes,
                             hipMemcpyDeviceToDevice, *stream));
}

void CudaExecutor::_copy_from(void* dst_ptr, const DeviceExecutor& src_exec,
                              const void* src_ptr, size_t num_bytes) const
{
  if (num_bytes > 0) {
    if (typeid(src_exec) == typeid(ReferenceExecutor)) {
      CHECK_CUDA(
          hipMemcpy(dst_ptr, src_ptr, num_bytes, hipMemcpyHostToDevice));
    } else if (typeid(src_exec) == typeid(CudaExecutor)) {
      CHECK_CUDA(hipMemcpyPeer(dst_ptr, this->get_device_id(), src_ptr,
                                src_exec.get_device_id(), num_bytes));
    }
  }
}

void CudaExecutor::_copy_to(void* dst_ptr, const DeviceExecutor& dst_exec,
                            const void* src_ptr, size_t num_bytes) const
{
  if (num_bytes > 0) {
    if (typeid(dst_exec) == typeid(ReferenceExecutor)) {
      CHECK_CUDA(
          hipMemcpy(dst_ptr, src_ptr, num_bytes, hipMemcpyDeviceToHost));
    }
  }
}

void CudaExecutor::spmv_init(CSRSpMV<float>& op, CSRMatrix<float>& mat,
                             bool symmetric)
{
  op.init(mat.rows(), mat.cols(), mat.non_zeros(), mat.rowptr(), mat.colind(),
          mat.values(), symmetric, *this);
}

void CudaExecutor::spmv_init(CSRSpMV<double>& op, CSRMatrix<double>& mat,
                             bool symmetric)
{
  op.init(mat.rows(), mat.cols(), mat.non_zeros(), mat.rowptr(), mat.colind(),
          mat.values(), symmetric, *this);
}

void CudaExecutor::spmv_run(const CSRSpMV<float>& op,
                            const CSRMatrix<float>& mat, float alpha,
                            float* __restrict__ in, float beta,
                            float* __restrict__ out) const
{
  op.run(mat.rows(), mat.cols(), mat.non_zeros(), mat.rowptr(), mat.colind(),
         mat.values(), mat.diagonal(), alpha, in, beta, out, *this);
}

void CudaExecutor::spmv_run(const CSRSpMV<double>& op,
                            const CSRMatrix<double>& mat, double alpha,
                            double* __restrict__ in, double beta,
                            double* __restrict__ out) const
{
  op.run(mat.rows(), mat.cols(), mat.non_zeros(), mat.rowptr(), mat.colind(),
         mat.values(), mat.diagonal(), alpha, in, beta, out, *this);
}

void CudaExecutor::spmv_finalize(const CSRSpMV<float>& op) const
{
  op.finalize(*this);
}

void CudaExecutor::spmv_finalize(const CSRSpMV<double>& op) const
{
  op.finalize(*this);
}

void CudaExecutor::spmv_init(COOSpMV<float>& op, COOMatrix<float>& mat)
{
  op.init(mat.rows(), mat.cols(), mat.non_zeros(), mat.rowind(), mat.colind(),
          mat.values(), *this);
}

void CudaExecutor::spmv_init(COOSpMV<double>& op, COOMatrix<double>& mat)
{
  op.init(mat.rows(), mat.cols(), mat.non_zeros(), mat.rowind(), mat.colind(),
          mat.values(), *this);
}

void CudaExecutor::spmv_run(const COOSpMV<float>& op,
                            const COOMatrix<float>& mat, float alpha,
                            float* __restrict__ in, float beta,
                            float* __restrict__ out) const
{
  op.run(mat.rows(), mat.cols(), mat.non_zeros(), mat.rowind(), mat.colind(),
         mat.values(), alpha, in, beta, out, *this);
}

void CudaExecutor::spmv_run(const COOSpMV<double>& op,
                            const COOMatrix<double>& mat, double alpha,
                            double* __restrict__ in, double beta,
                            double* __restrict__ out) const
{
  op.run(mat.rows(), mat.cols(), mat.non_zeros(), mat.rowind(), mat.colind(),
         mat.values(), alpha, in, beta, out, *this);
}

void CudaExecutor::spmv_finalize(const COOSpMV<float>& op) const
{
  op.finalize(*this);
}

void CudaExecutor::spmv_finalize(const COOSpMV<double>& op) const
{
  op.finalize(*this);
}

void CudaExecutor::set_cuda_stream(hipStream_t stream)
{
  _stream = stream;
  CHECK_CUBLAS(hipblasSetStream(_cublas_handle, stream));
  CHECK_CUSPARSE(hipsparseSetStream(_cusparse_handle, stream));
}

void CudaExecutor::reset_cuda_stream()
{
  _stream = nullptr;
  CHECK_CUBLAS(hipblasSetStream(_cublas_handle, 0));
  CHECK_CUSPARSE(hipsparseSetStream(_cusparse_handle, 0));
}

template <typename T>
__global__ void gather_ghosts(const int N, const int* indices, const T* in,
                              T* out)
{
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < N) {
    out[gid] = in[indices[gid]];
  }
}

void CudaExecutor::gather_ghosts_run(int num_indices, const int32_t* indices,
                                     const float* in, float* out) const
{
  const int block_size = 128;
  const int num_blocks = (num_indices + block_size - 1) / block_size;
  dim3 dimBlock(block_size);
  dim3 dimGrid(num_blocks);
  if (_stream == nullptr) {
    gather_ghosts<<<dimGrid, dimBlock>>>(num_indices, indices, in, out);
    hipDeviceSynchronize();
  } else {
    gather_ghosts<<<dimGrid, dimBlock, 0, _stream>>>(num_indices, indices, in,
                                                     out);
    hipStreamSynchronize(_stream);
  }
}

void CudaExecutor::gather_ghosts_run(int num_indices, const int32_t* indices,
                                     const double* in, double* out) const
{
  const int block_size = 128;
  const int num_blocks = (num_indices + block_size - 1) / block_size;
  dim3 dimBlock(block_size);
  dim3 dimGrid(num_blocks);
  if (_stream == nullptr) {
    gather_ghosts<<<dimGrid, dimBlock>>>(num_indices, indices, in, out);
    hipDeviceSynchronize();
  } else {
    gather_ghosts<<<dimGrid, dimBlock, 0, _stream>>>(num_indices, indices, in,
                                                     out);
    hipStreamSynchronize(_stream);
  }
}

} // namespace spmv
